#include <stdio.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <helper.h>
#include <tensor.h>

using namespace std;

Tensor::Tensor(vector<int> s, string d): shape(s), device(d), data(nullptr){
    int size = Tensor::get_size();
    if (device=="CPU"){
        data = (float*)malloc(size * sizeof(float));
    }
    else{
        hipMalloc(&data, size*sizeof(float));
    }
}

Tensor::~Tensor(){
    if (device=="CPU"){
        free(data);
    }
    else{
        hipFree(data);
    }
}

void Tensor::cpu(){
    if (device=="GPU"){
        float* data_cpu = nullptr;
        int size = get_size();
        data_cpu = (float*)malloc(size*sizeof(float));
        hipMemcpy(data_cpu, data, size*sizeof(float), hipMemcpyDeviceToHost);
        hipFree(data);
        data = data_cpu;
        device = "CPU";
    }
}

void Tensor::gpu(){
    if (device=="CPU"){
        float* data_gpu = nullptr;
        int size = get_size();
        hipMalloc(&data_gpu, size*sizeof(float));
        hipMemcpy(data_gpu, data, size*sizeof(float), hipMemcpyHostToDevice);
        free(data);
        data = data_gpu;
        // hipMemcpy(data, data_gpu, size*sizeof(float), hipMemcpyDeviceToHost);
        device = "GPU";
    }
}

int Tensor::get_size(){
    int size = 1;
    for (int i=0; i<shape.size(); ++i){
        size *= shape[i];
    }
    return size;
}