#include <hipblas.h>
#include <vector>

void gemm_gpu(hipblasOperation_t trans_A, hipblasOperation_t trans_B, const int m, const int k, const int n, 
const float alpha, const float *A, const float *B, const float beta, float *C)
{
    int lda = m, ldb = k, ldc = m;
    if (trans_A == HIPBLAS_OP_T)
        lda = k;
    if (trans_B == HIPBLAS_OP_T)
        ldb = n;

    // Create a handle for CUBLAS
    hipblasHandle_t handle; 
    hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, trans_A, trans_B, m, n, k, &alpha, 
    A, lda, B, ldb, &beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);

    hipDeviceSynchronize();
}

int main(){
    std::vector<int> s1 = {1,2,3,4,5,6};
    std::vector<int> s2 = {7,8,9,10,11,12};
    int *C = new int[20];
    // Create a handle for CUBLAS
    hipblasHandle_t handle; 
    hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, trans_A, trans_B, m, n, k, &alpha, 
    A, lda, B, ldb, &beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);

    hipDeviceSynchronize();

    return 0;
}