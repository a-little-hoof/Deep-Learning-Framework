#include "stdio.h"
#include "hip/hip_runtime.h"

__global__ void hello( )
{
   printf("GPU!");
   printf("blockIdx.x=%d/%d blocks, threadIdx.x=%d/%d threads\n",
                        blockIdx.x,  gridDim.x,
                        threadIdx.x, blockDim.x);
}

int main()
{
   // hello<<< 1, 1025 >>>( );    // Error !!!
   hello<<< 1, 256 >>>( );    // Correct !!!
   hipError_t err = hipGetLastError();        // Get error code

   if ( err != hipSuccess )
   {
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      exit(-1);
   }

   printf("I am the CPU: Hello World ! \n");

   hipDeviceSynchronize();

   return 0;
}
