#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <string>
#include <tensor.h>
#include <layer.h>
#include <utils.h>

// fc layer forward and backward
// X: [N, C_in], Y: [N, C_out], W: [C_in, C_out], b: [C_out]
// backward function takes partial L / partial y and outputs parital L / partial W or partial L / partial X

void fc_forward(const Tensor& X, const Tensor& W, const Tensor& b, Tensor& Y){
    int batch_size = X.shape[0];
    int in_features = X.shape[1];
    int out_features = W.shape[1];

    // matrix product with gemm
    gemm_gpu(HIPBLAS_OP_N, HIPBLAS_OP_N, batch_size, out_features, in_features,  
        1.0, X.data, W.data, 0.0, Y.data);

    // add bias
    Tensor ones_(std::vector<int>{batch_size, 1}, "GPU");
    ones_.fill_(1.0);
    gemm_gpu(HIPBLAS_OP_N, HIPBLAS_OP_N, batch_size, out_features, 1,
        1.0, ones_.data, b.data, 1.0, Y.data);
}

// dY: [N, C_out], X: [N, C_in], W: [C_in, C_out]
// dW: [C_in, C_out], db: [C_out], dX: [N, C_in]
void fc_backward(const Tensor& dY, const Tensor& X, const Tensor& W, Tensor& dW, Tensor& dX){

    int batch_size = X.shape[0];
    int in_features = X.shape[1];
    int out_features = W.shape[1];

    // dW = X^T * dY
    gemm_gpu(HIPBLAS_OP_T, HIPBLAS_OP_N, in_features, out_features, batch_size,   
        1.0, X.data, dY.data, 0.0, dW.data);

    // dX = dY * W^T
    gemm_gpu(HIPBLAS_OP_N, HIPBLAS_OP_T, batch_size, in_features, out_features,  
        1.0, dY.data, W.data, 0.0, dX.data);
    

}

//conv_layer forward and backward
// X: [N, C_in, H, W], Y: [N, C_out, H_out, W_out], W: [C_out, C_in, 3, 3], b: [C_out]
// backward function takes partial L / partial y and outputs parital L / partial W or partial L / partial X
void conv_forward(const Tensor& X, const Tensor& W, Tensor& Y){

    int batch_size = X.shape[0];
    int C_in = X.shape[1];
    int height = X.shape[2];
    int width = X.shape[3];
    int C_out = Y.shape[1];
    int H_out = Y.shape[2];
    int W_out = Y.shape[3];
    int len = C_out * H_out * W_out;
    
    //iterate over batch, in each image of the batch, it is converted into a matrix
    //then we do a matrix product with transformed W
    //W is transformed through flatten: [C_out, C_in, 3, 3] -> [C_out, C_in * 3 * 3]
    for(int i = 0; i < batch_size; i++){

        //transform X to X_hat: [C_in, H_out, W_out] -> [C_in * 3 * 3, H_out * W_out]
        Tensor X_hat(std::vector<int>{C_in*3*3, H_out*W_out}, "GPU");

        //calculate the number of kernels we are going to launch,
        //each kernel is responsible for copying a single-channel kernel_w*kernel_h pixels
        //and formulate them into a column, or precisely, 1/3 column
        int num_kernels = C_in * height * width;
    
        im2col_gpu_kernel<<<CudaGetBlocks(num_kernels), kCudaThreadsNum>>>(
            num_kernels, X.data, 
            height, width, 3, 3, 1, 1,
            X_hat.data);

        // matrix product with gemm
        
        gemm_gpu(HIPBLAS_OP_N, HIPBLAS_OP_N, C_out, H_out*W_out, C_in*3*3, 
            1.0, W.data, X_hat.data, 0.0, Y.data + i * len);
    }
    hipDeviceSynchronize();
}

//function adapted from caffe
__global__ void im2col_gpu_kernel(const int n, const float* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    float* data_col) {

    //each index stands for a single-channel pixel in the output matrix
    CUDA_KERNEL_LOOP(index, n){
        //transform the index to the corresponding pixel in the col matrix: [C_in*kernel_h*kernel_w, h_out*w_out]
    
        const int h_index = index / width; //we first calculate the height index
        const int h_col = h_index % height; //because there are multiple channels, we take % to get the height index(h_col) in the current h*w area
        const int w_col = index % width; //calculate the width index of the img
        
        const int c_im = index / (height * width); //calculate the channel index of the img
        
        const int c_col = c_im * kernel_h * kernel_w; //calculate the channel index of the output matrix
        
        const int h_offset = h_col - pad_h;
        const int w_offset = w_col - pad_w; 
        //calculate the offset of the height and width in the input matrix
        //it should be similar to h_col and w_col, but we need to consider the padding and relocate the pointer to the correct position
        //which is the left-top corner of the kernel

        //data_col_ptr points to the current pixel in the output matrix
        float* data_col_ptr = data_col;
        data_col_ptr += (c_col * height + h_col) * width + w_col;

        //data_im_ptr points to the current pixel in the input matrix
        const float* data_im_ptr = data_im;
        data_im_ptr += (c_im * height + h_offset) * width + w_offset;

        //iterate over the kernel, copy the pixel value to the output matrix
        for (int i = 0; i < kernel_h; ++i) {
            for (int j = 0; j < kernel_w; ++j) {
                int h_im = h_offset + i;
                int w_im = w_offset + j;
                *data_col_ptr =
                (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width) ?
                data_im_ptr[i * width + j] : 0;
                data_col_ptr += height * width; //becaues we are formulating a column, we need to move the pointer to the next pixel in the column
            }
        }
    }
}

void conv_backward(const Tensor& dY, const Tensor& X, const Tensor& W, Tensor& dW, Tensor& dX){

}

// __global__ void col2im_gpu_kernel(const int n, const float* data_col,
//     const int height, const int width, const int channels,
//     const int kernel_h, const int kernel_w,
//     const int pad_h, const int pad_w,
//     const int stride_h, const int stride_w,
//     const int dilation_h, const int dilation_w,
//     const int height_col, const int width_col,
//     float* data_im) {
//   CUDA_KERNEL_LOOP(index, n) {
//     float val = 0;
//     const int w_im = index % width + pad_w;
//     const int h_im = (index / width) % height + pad_h;
//     const int c_im = index / (width * height);
//     int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
//     int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
//     // compute the start and end of the output
//     const int w_col_start =
//         (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
//     const int w_col_end = min(w_im / stride_w + 1, width_col);
//     const int h_col_start =
//         (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
//     const int h_col_end = min(h_im / stride_h + 1, height_col);
//     // TODO: use LCM of stride and dilation to avoid unnecessary loops
//     for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
//       for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
//         int h_k = (h_im - h_col * stride_h);
//         int w_k = (w_im - w_col * stride_w);
//         if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
//           h_k /= dilation_h;
//           w_k /= dilation_w;
//           int data_col_index = (((c_im * kernel_h + h_k) * kernel_w + w_k) *
//                                 height_col + h_col) * width_col + w_col;
//           val += data_col[data_col_index];
//         }
//       }
//     }
//     data_im[index] = val;
//   }
// }

// void col2im_gpu(const float* data_col, const int channels,
//     const int height, const int width, const int kernel_h, const int kernel_w,
//     const int pad_h, const int pad_w, const int stride_h,
//     const int stride_w, const int dilation_h, const int dilation_w,
//     float* data_im) {
//   int height_col = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) /
//       stride_h + 1;
//   int width_col = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) /
//       stride_w + 1;
//   int num_kernels = channels * height * width;
//   // To avoid involving atomic operations, we will launch one kernel per
//   // bottom dimension, and then in the kernel add up the top dimensions.
//   // NOLINT_NEXT_LINE(whitespace/operators)
//   col2im_gpu_kernel<<<CAFFE_GET_BLOCKS(num_kernels),
//                              CAFFE_CUDA_NUM_THREADS>>>(
//       num_kernels, data_col, height, width, channels, kernel_h, kernel_w,
//       pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w,
//       height_col, width_col, data_im);
//   CUDA_POST_KERNEL_CHECK;
// }



void gemm_gpu(hipblasOperation_t trans_A, hipblasOperation_t trans_B, const int m, const int n, const int k,  
const float alpha, const float *A, const float *B, const float beta, float *C)
{
    int lda = k, ldb = n, ldc = n;
    if (trans_A == HIPBLAS_OP_T)
        lda = m;
    if (trans_B == HIPBLAS_OP_T)
        ldb = k;

    // printf("m=%d, k=%d, n=%d\n", m, k, n);
    // printf("lda=%d, ldb=%d, ldc=%d\n", lda, ldb, ldc);

    // Create a handle for CUBLAS
    hipblasHandle_t handle; 
    hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasSgemm(handle, trans_B, trans_A, n, m, k, &alpha, 
    B, ldb, A, lda, &beta, C, ldc);
    // Destroy the handle
    hipblasDestroy(handle);

    hipDeviceSynchronize();
}